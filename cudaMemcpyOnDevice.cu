#include "hip/hip_runtime.h"
#include "./common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000*10

__global__ void Kernel_Memcpy(int* dst, int* src){
  int thid = threadIdx.x;
  int blid = blockIdx.x;
  int i;

  if(thid < 1 && blid < 1){

    printf("device src:\n");
    for ( i = 0; i < N; i++) {
      printf("%d ", src[i]);
    }
    printf("\n");

    printf("device dst:\n");
    for ( i = 0; i < N; i++) {
      printf("%d ", dst[i]);
    }
    printf("\n");
    printf("hipMemcpyAsync()\n");
    hipMemcpyAsync(dst, src, sizeof(int) * N ,hipMemcpyDeviceToDevice);
    // for ( i = 0; i < N; i++) {
    //   dst[i] = src[i];
    // }
    printf("device src:\n");
    for ( i = 0; i < N; i++) {
      printf("%d ", src[i]);
    }
    printf("\n");

    printf("device dst:\n");
    for ( i = 0; i < N; i++) {
      printf("%d ", dst[i]);
    }
    printf("\n");
  }
  else{}
}//end-kernel

__global__ void Kernel_print_array(int *array){
  int i;
  printf("N: %d\n",N );
  printf("%s\n","device array" );
  for ( i = 0; i < N; i++) {
    printf("%d ", array[i]);
  }
  printf("\n");
}//end-kernel

int main(void){

  int *h_src, *h_dst;
  int *d_src, *d_dst;
  //----set up device START-----
  int dev_num =0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,dev_num);
  printf("Using Device %d:%s\n",dev_num,deviceProp.name);
  hipSetDevice(dev_num);
  //----set up device END-----

  h_src = (int*)malloc(sizeof(int) * N);
  h_dst = (int*)malloc(sizeof(int) * N);

  memset(h_src, 0, sizeof(int) * N);
  memset(h_dst, 0, sizeof(int) * N);

  printf("host src:\n");
  for (size_t i = 0; i < N; i++) {
    h_src[i] = i+1;
    printf("%d ", h_src[i]);
  }
  printf("\n");

  printf("host dst:\n");
  for (size_t i = 0; i < N; i++) {
    printf("%d ", h_dst[i]);
  }
  printf("\n");

  hipMalloc((int**)&d_src, sizeof(int)*N);
  hipMalloc((int**)&d_dst, sizeof(int)*N);
  hipMemcpy(d_src, h_src, sizeof(int)*N, hipMemcpyHostToDevice);
  printf("Kernel_Memcpy<<<>>>()\n");
  Kernel_Memcpy<<<1,1>>>(d_dst, d_src);
  hipMemcpy(h_dst, d_dst, sizeof(int)*N, hipMemcpyDeviceToHost);

  printf("host src:\n");
  for (size_t i = 0; i < N; i++) {
    printf("%d ", h_src[i]);
  }
  printf("\n");

  printf("host dst:\n");
  for (size_t i = 0; i < N; i++) {
    printf("%d ", h_dst[i]);
  }
  printf("\n");
  CHECK(hipDeviceReset());
  return EXIT_SUCCESS;
}
